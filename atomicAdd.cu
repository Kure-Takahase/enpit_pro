
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void add1(int *g_A){
	atomicAdd(&g_A[0], 1);
}

int main()
{
	int *h_A;
	int *d_A;

	h_A = (int*)malloc(sizeof(int));

	hipMalloc((void**)&d_A, sizeof(int));
	hipMemset(d_A,0,sizeof(int));

	add1<<<1024, 1024>>>(d_A);

	hipMemcpy(h_A,d_A,sizeof(int),hipMemcpyDeviceToHost);

	printf("%d\n",h_A[0]);

	hipFree(d_A);
	free(h_A);
	
	return 0;
}