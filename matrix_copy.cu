
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

#define N 1024 //N*N行列

__global__ void matrix_copy(int *g_A, int *g_B)
{
	int gx = blockIdx.x*blockDim.x + threadIdx.x;
	int gy = blockIdx.y*blockDim.y + threadIdx.y;
	g_B[gy * N + gx] = g_A[gy * N + gx];
}

int main()
{
	int *h_A, *h_B, *d_A, *d_B;
	
	h_A = (int*)malloc(N*N*sizeof(int));
	h_B = (int*)malloc(N*N*sizeof(int));
	
	hipMalloc(&d_A, N*N*sizeof(int));
	hipMalloc(&d_B, N*N*sizeof(int));
	
	for(int i=0; i<N*N; i++) h_A[i] = i;
	
	hipMemcpy(d_A, h_A, N*N*sizeof(int), hipMemcpyHostToDevice);
	
	dim3 grid(32, 32);
	dim3 block(32, 32);
	
	matrix_copy<<< grid, block >>>(d_A, d_B);
	
	hipMemcpy(h_B, d_B, N*N*sizeof(int), hipMemcpyDeviceToHost);
	
	int flag = 0;
	for(int y=0;y<N;y++)
		for(int x=0;x<N;x++)
			if(h_A[y*N+x]!=h_B[y*N+x]){
				flag = 1;
				break;
			}
	
	if(flag==0)
		printf("OK\n");
	else
		printf("NG\n");
	

	free(h_A);
	free(h_B);
	
	hipFree(d_A);
	hipFree(d_B);
}