#include "hip/hip_runtime.h"
#include<stdio.h>
#include"io_wrapper.h"

__global__ void convert(const unsigned char* g_in, unsigned char* g_out)
{
	int gy = blockIdx.y*blockDim.y + threadIdx.y;
	int gx = blockIdx.x*blockDim.x + threadIdx.x;

	g_out[gy*512 + gx] = (255 - g_in[gy*512 + gx]);
}


int main(){
	unsigned char *h_in=NULL, *h_out=NULL;
	int H, W;
	FILE *fp0, *fp1;

	if(	((fp0 = fopen("lena.pgm", "r")) == NULL)
		|| ((fp1 = fopen("lena_converted.pgm", "w")) == NULL))
	{
		printf("file open error!\n");
		exit(1);
	}
	
	h_in = pgm_read(fp0, &H, &W);
	h_out = (unsigned char *)malloc(sizeof(unsigned char)*512*512);
	
	unsigned char *d_in, *d_out;
	
	hipMalloc(&d_in, sizeof(unsigned char)*512*512);
	hipMalloc(&d_out, sizeof(unsigned char)*512*512);
	
	hipMemcpy(d_in, h_in, sizeof(unsigned char)*512*512, hipMemcpyHostToDevice);
	
	dim3 grid(16, 16);
	dim3 block(32, 32);
	
	convert<<<grid, block>>>(d_in, d_out);
	
	hipMemcpy(h_out, d_out, sizeof(unsigned char)*512*512, hipMemcpyDeviceToHost);
	
	hipFree(d_in);
	hipFree(d_out);
	
	pgm_write(h_out, fp1, H, W);
	
	fclose(fp0);
	fclose(fp1);
	
	free(h_in);
	free(h_out);
	
	return 0;
}