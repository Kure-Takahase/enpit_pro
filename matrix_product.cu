
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 1024

__global__ void matrix_product(int *g_A, int *g_B, int *g_C) {
	int gx = threadIdx.x + blockIdx.x * 32;
	int gy = threadIdx.y + blockIdx.y * 32;
	int c = 0;
	int k;
	for (k = 0; k < N; k++) {
		c += g_A[k + gy*N] * g_B[gx + k*N];
	}
	g_C[gx + gy*N] = c;
}


int main() {
	int i;
	int *h_A, *h_B, *h_C, *d_A, *d_B, *d_C;
	
	h_A = (int*)malloc(sizeof(int)*N*N);
	h_B = (int*)malloc(sizeof(int)*N*N);
	h_C = (int*)malloc(sizeof(int)*N*N);
	
	hipMalloc(&d_A, sizeof(int)*N*N);
	hipMalloc(&d_B, sizeof(int)*N*N);
	hipMalloc(&d_C, sizeof(int)*N*N);
	
	for (i = 0; i < N*N; i++) {
		h_A[i] = h_B[i] = 1;
	}
	
	hipMemcpy(d_A, h_A, sizeof(int)*N*N, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, sizeof(int)*N*N, hipMemcpyHostToDevice);
	hipMemset(d_C, 0, sizeof(int)*N*N);
	
	dim3 grid(32, 32);
	dim3 block(32, 32);
	

	matrix_product<<< grid, block >>> (d_A, d_B, d_C);
	
	hipMemcpy(h_C, d_C, sizeof(int)*N*N, hipMemcpyDeviceToHost);
	
	int flag = 0;
		for(int y=0; y<N; y++){
			for(int x=0; x<N; x++){
				int c = 0;
				for(int k=0; k<N; k++){
					c += h_A[y*N + k] * h_B[k*N + x];
				}
				if(h_C[y*N + x] != c){
					flag = 1;
				}
			}
		}
	if(flag==0)
		printf("OK\n");
	else
		printf("NG\n");
		
	free(h_A);
	free(h_B);
	free(h_C);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
}