#include "hip/hip_runtime.h"
#include<stdio.h>
#include"io_wrapper.h"

//Laplacian Filter 3x3
__device__ const float d_f[3][3] = { {1.0f, 1.0f, 1.0f},
						{1.0f, -8.0f, 1.0f},
						{1.0f, 1.0f, 1.0f} };

__global__ void filter(const unsigned char* g_in, unsigned char* g_out) {
	int gy = blockIdx.y*blockDim.y + threadIdx.y;
	int gx = blockIdx.x*blockDim.x + threadIdx.x;
	
	if(gy == 0 || gx == 0 || gy == 511 || gx == 511){
		g_out[gy*512 + gx] = 0;
	}else{
		float val = 0.0f;
		for(int y=-1;y <= 1;++y){
			for(int x=-1;x <= 1;++x){
				val += g_in[(gy+y)*512+(gx+x)] * d_f[y+1][x+1];
			}
		}
		val = fabsf(val);
		g_out[gy*512 + gx] = (val > 255 ? 255 : (unsigned char)val);
	}
}

int main(){
	unsigned char *h_in=NULL, *h_out=NULL;
	int H, W;
	FILE *fp0, *fp1;

	if(	((fp0 = fopen("lena.pgm", "r")) == NULL)
		|| ((fp1 = fopen("lena_filtered.pgm", "w")) == NULL)) {
		printf("file open error!\n");
		exit(1);
	}

	h_in = pgm_read(fp0, &H, &W);
	h_out = (unsigned char *)malloc(sizeof(unsigned char)*512*512);
	
	unsigned char *d_in, *d_out;
	
	hipMalloc(&d_in, sizeof(unsigned char)*512*512);
	hipMalloc(&d_out, sizeof(unsigned char)*512*512);
	
	hipMemcpy(d_in, h_in, sizeof(unsigned char)*512*512, hipMemcpyHostToDevice);
	
	dim3 grid(16, 16);
	dim3 block(32, 32);
	
	filter<<<grid, block>>>(d_in, d_out);
	
	hipMemcpy(h_out, d_out, sizeof(unsigned char)*512*512, hipMemcpyDeviceToHost);
	
	hipFree(d_in);
	hipFree(d_out);
	
	pgm_write(h_out, fp1, 512, 512);
	
	fclose(fp0);
	fclose(fp1);
	
	free(h_in);
	free(h_out);
	
	return 0;
}