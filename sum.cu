
#include <hip/hip_runtime.h>
#include<stdio.h>

#define N (1024*1024)

__global__ void local_sum(int *g_A, int *g_B){
	__shared__ int s_A[1024];
	
	s_A[threadIdx.x]=g_A[threadIdx.x+1024*blockIdx.x];
	__syncthreads();
	for(int i=512;i>0;i>>=1){
		if(threadIdx.x<i){
			s_A[threadIdx.x]+=s_A[threadIdx.x+i];
		}
		__syncthreads();
	}

	if(threadIdx.x==0){
		g_B[blockIdx.x]=s_A[0];
	}
}
int main()
{
	int *h_A, *h_C;
	int *d_A, *d_B, *d_C;
	int ans;

	h_A = (int*)malloc(N*sizeof(int));
	h_C = (int*)malloc(sizeof(int));

	ans = 0;
	for (int i = 0; i < N; i++){
		h_A[i] = 1;
		ans += h_A[i];
	}

	hipMalloc((void**)&d_A, N*sizeof(int));
	hipMalloc((void**)&d_B, 1024*sizeof(int));
	hipMalloc((void**)&d_C, sizeof(int));

	hipMemcpy(d_A,h_A,N*sizeof(int),hipMemcpyHostToDevice);
	local_sum<<<1024, 1024>>>(d_A, d_B);
	local_sum<<< 1, 1024>>>(d_B, d_C);
	hipMemcpy(h_C,d_C,sizeof(int),hipMemcpyDeviceToHost);

	printf("%d %d\n",ans,h_C[0]);

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	free(h_A);
	free(h_C);
}