
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void kernel(){
	int gId = blockIdx.x * blockDim.x + threadIdx.x;
	printf("bId=%d,tId=%d,gId=%d¥n", blockIdx.x, threadIdx.x, gId);
}

int main(void){
	kernel<<<3, 4>>>();
	hipDeviceSynchronize();
	return 0;
}