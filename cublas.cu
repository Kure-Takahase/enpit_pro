
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<hipblas.h>

#define N 1024

int main(){
	float *A, *B, *C;
	float *g_A, *g_B, *g_C;
	
	const float alpha = 1.0f;
	const float beta = 0.0f;
	
	A=(float*)malloc(sizeof(float)*N*N);
	B=(float*)malloc(sizeof(float)*N*N);
	C=(float*)malloc(sizeof(float)*N*N);
	for(int i=0;i<N*N;i++){
		A[i]=1.0f; B[i]=1.0f; C[i]=-1.0f;
	}
	
	hipMalloc((void**)&g_A,sizeof(float)*N*N);
	hipMalloc((void**)&g_B,sizeof(float)*N*N);
	hipMalloc((void**)&g_C,sizeof(float)*N*N);
	
	hipMemcpy(g_A,A,sizeof(float)*N*N,hipMemcpyHostToDevice);
	hipMemcpy(g_B,B,sizeof(float)*N*N,hipMemcpyHostToDevice);
	hipMemcpy(g_C,C,sizeof(float)*N*N,hipMemcpyHostToDevice);
	
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N,
				&alpha, g_A, N, g_B, N, &beta, g_C, N);
	hipblasDestroy(handle);
	hipMemcpy(C,g_C,sizeof(float)*N*N,hipMemcpyDeviceToHost);
}