
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 32

__global__ void kernel(int* input, int* output){
	for(int i=0; i<N; i++)
		output[i] = 2 * input[i];
}
int main(void){
	int *h_input, *h_output;
	int *d_input, *d_output;
	
	h_input = (int*)malloc(N*sizeof(int));
	h_output = (int*)malloc(N*sizeof(int));
	
	hipMalloc((void**)&d_input, N*sizeof(int));
	hipMalloc((void**)&d_output, N*sizeof(int));
	
	for(int i=0; i<N; i++) h_input[i] = i+1;
	
	hipMemcpy(d_input, h_input, N*sizeof(int), hipMemcpyHostToDevice);
	
	kernel<<<1, 1>>> (d_input, d_output);
	
	hipDeviceSynchronize();
	
	hipMemcpy(h_output, d_output, N*sizeof(int), hipMemcpyDeviceToHost);
	

	for(int i=0; i<N; i++) printf("%d -> %d\n", h_input[i], h_output[i]);
	
	free(h_input);
	free(h_output);
	hipFree(d_input);
	hipFree(d_output);

	return 0;
}